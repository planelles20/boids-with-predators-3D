#include "hip/hip_runtime.h"
//-----------------------------------------------------------------------
// Code file of SistemSPH class which simulate boids model using
// Smoothed-particle hydrodynamics (SPH) method.
//
// Licensing: This code is distributed under the Apache License 2.0
// Author: Carlos Planelles Alemany, planelles20(at)gmail(dot)com
//-----------------------------------------------------------------------

#include "systemSPH.h"

////////////////////////////////  kernels  ////////////////////////////////////

__global__ void clearGridIndices_kernel(intvec2 *d_inidices, int numCells) {
    unsigned int i = threadIdx.x + blockDim.x * blockIdx.x;
    if(i < numCells){
        d_inidices[i].init = 0;
        d_inidices[i].end  = 0;
    }
}

__global__ void builtGridIncices_kernel(intvec2 *gridIdx, particle *d_particle, int numParticles){
    unsigned int i = threadIdx.x + blockDim.x * blockIdx.x;
    if(i < numParticles){
        if( i==0 && d_particle[i].cellIdx != d_particle[i+1].cellIdx){
            gridIdx[d_particle[i].cellIdx].init = 0;
        }

        if(i!=0 && d_particle[i].cellIdx != d_particle[i-1].cellIdx){
            gridIdx[d_particle[i].cellIdx].init = i;
        }
        if(i!=(numParticles-1) && d_particle[i].cellIdx != d_particle[i+1].cellIdx){
            gridIdx[d_particle[i].cellIdx].end = i;
        }
        if(i==(numParticles-1) && d_particle[i].cellIdx == d_particle[i-1].cellIdx){
            gridIdx[d_particle[i].cellIdx].end = i;
        }
    }
}

__global__ void posParticleCell_kernel(particle *d_particle, int numParticles, int xMesh, int yMesh, int zMesh){
    unsigned int i = threadIdx.x + blockDim.x * blockIdx.x;
    if(i<numParticles){
        int ii = int(d_particle[i].x*float(xMesh));
        int jj = int(d_particle[i].y*float(yMesh));
        int kk = int(d_particle[i].z*float(zMesh));
        if(ii == xMesh) ii--;
        if(jj == yMesh) jj--;
        if(kk == zMesh) kk--;
        d_particle[i].cellIdx = ii+jj*xMesh+kk*xMesh*yMesh;
    }
}

__global__ void bitonic_sort_kernel(particle *dev_values, int j, int k) {
  unsigned int i, ixj; /* Sorting partners: i and ixj */
  i = threadIdx.x + blockDim.x * blockIdx.x;
  ixj = i^j;

  /* The threads with the lowest ids sort the array. */
  if ((ixj)>i) {
    if ((i&k)==0) {
      /* Sort ascending */
      if (dev_values[i].cellIdx>dev_values[ixj].cellIdx) {
        /* exchange(i,ixj) particles; */
        //cell
        particle temp = dev_values[i];
        dev_values[i] = dev_values[ixj];
        dev_values[ixj] = temp;
      }
    }
    if ((i&k)!=0) {
      /* Sort descending */
      if (dev_values[i].cellIdx<dev_values[ixj].cellIdx) {
        /* exchange(i,ixj); */
        particle temp = dev_values[i];
        dev_values[i] = dev_values[ixj];
        dev_values[ixj] = temp;
      }
    }
  }
}

__global__ void calcul_kernel(particle *d_particle, intvec2 *d_inidices,
                              calcStruct *d_calc, int numParticles,
                              int xMesh, int yMesh, int zMesh){

    unsigned int i = threadIdx.x + blockDim.x * blockIdx.x;
    if(i < numParticles){
        // number of near boids
        int sepa_n = 0;
        int alig_n = 0;
        int cohe_n = 0;
        int pred_n = 0;

        float sepa_x = 0.0,  sepa_y = 0.0,  sepa_z = 0.0;
        float alig_x = 0.0,  alig_y = 0.0,  alig_z = 0.0;
        float cohe_x = 0.0,  cohe_y = 0.0,  cohe_z = 0.0;
        float pred_x = 0.0,  pred_y = 0.0,  pred_z = 0.0;

        float cohe_r = 1.0/float(xMesh);
        float alig_r = 0.7*cohe_r;
        float sepa_r = 0.4*cohe_r;

        if(d_particle[i].tag > 0){
            alig_r = 0.2*cohe_r;
            sepa_r = 0.1*cohe_r;
        }

        // look own cell
        for(int ii=d_inidices[d_particle[i].cellIdx].init; ii<d_inidices[d_particle[i].cellIdx].end; ++ii){
            float dist = sqrt(pow(d_particle[ii].x-d_particle[i].x, 2)+
                              pow(d_particle[ii].y-d_particle[i].y, 2)+
                              pow(d_particle[ii].z-d_particle[i].z, 2));

            if(d_particle[ii].tag > 0 && d_particle[i].tag < 1 && i!=ii){
                pred_n++;
                pred_x += (d_particle[i].x-d_particle[ii].x)/dist;
                pred_y += (d_particle[i].y-d_particle[ii].y)/dist;
                pred_z += (d_particle[i].z-d_particle[ii].z)/dist;
            }
            else if(dist < sepa_r && i!=ii){
               sepa_n++;
               sepa_x += (d_particle[i].x-d_particle[ii].x)/dist;
               sepa_y += (d_particle[i].y-d_particle[ii].y)/dist;
               sepa_z += (d_particle[i].z-d_particle[ii].z)/dist;
            }
            else if(dist < alig_r && i!=ii){
               alig_n++;
               alig_x += d_particle[ii].vx;
               alig_y += d_particle[ii].vy;
               alig_z += d_particle[ii].vz;
            }
            else if(dist < cohe_r && i!=ii){
               cohe_n++;
               cohe_x += d_particle[ii].x;
               cohe_y += d_particle[ii].y;
               cohe_z += d_particle[ii].z;
            }
        }
        /// looking near cells
        // x+1
        if(d_particle[i].cellIdx+1 < xMesh*yMesh*zMesh){
            for(int ii=d_inidices[d_particle[i].cellIdx+1].init; ii<d_inidices[d_particle[i].cellIdx+1].end; ++ii){
                float dist = sqrt(pow(d_particle[ii].x-d_particle[i].x, 2)+
                                  pow(d_particle[ii].y-d_particle[i].y, 2)+
                                  pow(d_particle[ii].z-d_particle[i].z, 2));

                if(d_particle[ii].tag > 0 && d_particle[i].tag < 1){
                    pred_n++;
                    pred_x += (d_particle[i].x-d_particle[ii].x)/dist;
                    pred_y += (d_particle[i].y-d_particle[ii].y)/dist;
                    pred_z += (d_particle[i].z-d_particle[ii].z)/dist;
                }
                else if(dist < sepa_r){
                   sepa_n++;
                   sepa_x += (d_particle[i].x-d_particle[ii].x)/dist;
                   sepa_y += (d_particle[i].y-d_particle[ii].y)/dist;
                   sepa_z += (d_particle[i].z-d_particle[ii].z)/dist;
                }
                else if(dist < alig_r){
                   alig_n++;
                   alig_x += d_particle[ii].vx;
                   alig_y += d_particle[ii].vy;
                   alig_z += d_particle[ii].vz;
                }
                else if(dist < cohe_r){
                   cohe_n++;
                   cohe_x += d_particle[ii].x;
                   cohe_y += d_particle[ii].y;
                   cohe_z += d_particle[ii].z;
                }
            }
        }
        // x-1
        if(d_particle[i].cellIdx-1 > 0){
            for(int ii=d_inidices[d_particle[i].cellIdx-1].init; ii<d_inidices[d_particle[i].cellIdx-1].end; ++ii){
                float dist = sqrt(pow(d_particle[ii].x-d_particle[i].x, 2)+
                                  pow(d_particle[ii].y-d_particle[i].y, 2)+
                                  pow(d_particle[ii].z-d_particle[i].z, 2));

                if(d_particle[ii].tag > 0 && d_particle[i].tag < 1){
                    pred_n++;
                    pred_x += (d_particle[i].x-d_particle[ii].x)/dist;
                    pred_y += (d_particle[i].y-d_particle[ii].y)/dist;
                    pred_z += (d_particle[i].z-d_particle[ii].z)/dist;
                }
                else if(dist < sepa_r){
                   sepa_n++;
                   sepa_x += (d_particle[i].x-d_particle[ii].x)/dist;
                   sepa_y += (d_particle[i].y-d_particle[ii].y)/dist;
                   sepa_z += (d_particle[i].z-d_particle[ii].z)/dist;
                }
                else if(dist < alig_r){
                   alig_n++;
                   alig_x += d_particle[ii].vx;
                   alig_y += d_particle[ii].vy;
                   alig_z += d_particle[ii].vz;
                }
                else if(dist < cohe_r){
                   cohe_n++;
                   cohe_x += d_particle[ii].x;
                   cohe_y += d_particle[ii].y;
                   cohe_z += d_particle[ii].z;
                }
            }
        }
        // y+1
        if(d_particle[i].cellIdx+1*xMesh < xMesh*yMesh*zMesh){
            for(int ii=d_inidices[d_particle[i].cellIdx+1*xMesh].init; ii<d_inidices[d_particle[i].cellIdx+1*xMesh].end; ++ii){
                float dist = sqrt(pow(d_particle[ii].x-d_particle[i].x, 2)+
                                  pow(d_particle[ii].y-d_particle[i].y, 2)+
                                  pow(d_particle[ii].z-d_particle[i].z, 2));

                if(d_particle[ii].tag > 0 && d_particle[i].tag < 1){
                    pred_n++;
                    pred_x += (d_particle[i].x-d_particle[ii].x)/dist;
                    pred_y += (d_particle[i].y-d_particle[ii].y)/dist;
                    pred_z += (d_particle[i].z-d_particle[ii].z)/dist;
                }
                else if(dist < sepa_r){
                   sepa_n++;
                   sepa_x += (d_particle[i].x-d_particle[ii].x)/dist;
                   sepa_y += (d_particle[i].y-d_particle[ii].y)/dist;
                   sepa_z += (d_particle[i].z-d_particle[ii].z)/dist;
                }
                else if(dist < alig_r){
                   alig_n++;
                   alig_x += d_particle[ii].vx;
                   alig_y += d_particle[ii].vy;
                   alig_z += d_particle[ii].vz;
                }
                else if(dist < cohe_r){
                   cohe_n++;
                   cohe_x += d_particle[ii].x;
                   cohe_y += d_particle[ii].y;
                   cohe_z += d_particle[ii].z;
                }
            }
        }
        // y-1
        if(d_particle[i].cellIdx-1*xMesh > 0){
            for(int ii=d_inidices[d_particle[i].cellIdx-1*xMesh].init; ii<d_inidices[d_particle[i].cellIdx-1*xMesh].end; ++ii){
                float dist = sqrt(pow(d_particle[ii].x-d_particle[i].x, 2)+
                                  pow(d_particle[ii].y-d_particle[i].y, 2)+
                                  pow(d_particle[ii].z-d_particle[i].z, 2));

                if(d_particle[ii].tag > 0 && d_particle[i].tag < 1){
                    pred_n++;
                    pred_x += (d_particle[i].x-d_particle[ii].x)/dist;
                    pred_y += (d_particle[i].y-d_particle[ii].y)/dist;
                    pred_z += (d_particle[i].z-d_particle[ii].z)/dist;
                }
                else if(dist < sepa_r){
                   sepa_n++;
                   sepa_x += (d_particle[i].x-d_particle[ii].x)/dist;
                   sepa_y += (d_particle[i].y-d_particle[ii].y)/dist;
                   sepa_z += (d_particle[i].z-d_particle[ii].z)/dist;
                }
                else if(dist < alig_r){
                   alig_n++;
                   alig_x += d_particle[ii].vx;
                   alig_y += d_particle[ii].vy;
                   alig_z += d_particle[ii].vz;
                }
                else if(dist < cohe_r){
                   cohe_n++;
                   cohe_x += d_particle[ii].x;
                   cohe_y += d_particle[ii].y;
                   cohe_z += d_particle[ii].z;
                }
            }
        }
        // z+1
        if(d_particle[i].cellIdx+1*xMesh*yMesh < xMesh*yMesh*zMesh){
            for(int ii=d_inidices[d_particle[i].cellIdx+1*xMesh*yMesh].init; ii<d_inidices[d_particle[i].cellIdx+1*xMesh*yMesh].end; ++ii){
                float dist = sqrt(pow(d_particle[ii].x-d_particle[i].x, 2)+
                                  pow(d_particle[ii].y-d_particle[i].y, 2)+
                                  pow(d_particle[ii].z-d_particle[i].z, 2));

                if(d_particle[ii].tag > 0 && d_particle[i].tag < 1){
                    pred_n++;
                    pred_x += (d_particle[i].x-d_particle[ii].x)/dist;
                    pred_y += (d_particle[i].y-d_particle[ii].y)/dist;
                    pred_z += (d_particle[i].z-d_particle[ii].z)/dist;
                }
                else if(dist < sepa_r){
                   sepa_n++;
                   sepa_x += (d_particle[i].x-d_particle[ii].x)/dist;
                   sepa_y += (d_particle[i].y-d_particle[ii].y)/dist;
                   sepa_z += (d_particle[i].z-d_particle[ii].z)/dist;
                }
                else if(dist < alig_r){
                   alig_n++;
                   alig_x += d_particle[ii].vx;
                   alig_y += d_particle[ii].vy;
                   alig_z += d_particle[ii].vz;
                }
                else if(dist < cohe_r){
                   cohe_n++;
                   cohe_x += d_particle[ii].x;
                   cohe_y += d_particle[ii].y;
                   cohe_z += d_particle[ii].z;
                }
            }
        }
        // z-1
        if(d_particle[i].cellIdx-1*xMesh*yMesh > 0){
            for(int ii=d_inidices[d_particle[i].cellIdx-1*xMesh*yMesh].init; ii<d_inidices[d_particle[i].cellIdx-1*xMesh*yMesh].end; ++ii){
                float dist = sqrt(pow(d_particle[ii].x-d_particle[i].x, 2)+
                                  pow(d_particle[ii].y-d_particle[i].y, 2)+
                                  pow(d_particle[ii].z-d_particle[i].z, 2));

                if(d_particle[ii].tag > 0 && d_particle[i].tag < 1){
                    pred_n++;
                    pred_x += (d_particle[i].x-d_particle[ii].x)/dist;
                    pred_y += (d_particle[i].y-d_particle[ii].y)/dist;
                    pred_z += (d_particle[i].z-d_particle[ii].z)/dist;
                }
                else if(dist < sepa_r){
                   sepa_n++;
                   sepa_x += (d_particle[i].x-d_particle[ii].x)/dist;
                   sepa_y += (d_particle[i].y-d_particle[ii].y)/dist;
                   sepa_z += (d_particle[i].z-d_particle[ii].z)/dist;
                }
                else if(dist < alig_r){
                   alig_n++;
                   alig_x += d_particle[ii].vx;
                   alig_y += d_particle[ii].vy;
                   alig_z += d_particle[ii].vz;
                }
                else if(dist < cohe_r){
                   cohe_n++;
                   cohe_x += d_particle[ii].x;
                   cohe_y += d_particle[ii].y;
                   cohe_z += d_particle[ii].z;
                }
            }
        }
        // x+1, y+1
        if(d_particle[i].cellIdx+1+1*xMesh > 0 && d_particle[i].cellIdx+1+1*xMesh < xMesh*yMesh*zMesh){
            for(int ii=d_inidices[d_particle[i].cellIdx+1+1*xMesh].init; ii<d_inidices[d_particle[i].cellIdx+1+1*xMesh].end; ++ii){
                float dist = sqrt(pow(d_particle[ii].x-d_particle[i].x, 2)+
                                  pow(d_particle[ii].y-d_particle[i].y, 2)+
                                  pow(d_particle[ii].z-d_particle[i].z, 2));

                if(d_particle[ii].tag > 0 && d_particle[i].tag < 1){
                    pred_n++;
                    pred_x += (d_particle[i].x-d_particle[ii].x)/dist;
                    pred_y += (d_particle[i].y-d_particle[ii].y)/dist;
                    pred_z += (d_particle[i].z-d_particle[ii].z)/dist;
                }
                else if(dist < sepa_r){
                   sepa_n++;
                   sepa_x += (d_particle[i].x-d_particle[ii].x)/dist;
                   sepa_y += (d_particle[i].y-d_particle[ii].y)/dist;
                   sepa_z += (d_particle[i].z-d_particle[ii].z)/dist;
                }
                else if(dist < alig_r){
                   alig_n++;
                   alig_x += d_particle[ii].vx;
                   alig_y += d_particle[ii].vy;
                   alig_z += d_particle[ii].vz;
                }
                else if(dist < cohe_r){
                   cohe_n++;
                   cohe_x += d_particle[ii].x;
                   cohe_y += d_particle[ii].y;
                   cohe_z += d_particle[ii].z;
                }
            }
        }
        // x+1, y-1
        if(d_particle[i].cellIdx+1-1*xMesh > 0 && d_particle[i].cellIdx+1-1*xMesh < xMesh*yMesh*zMesh){
            for(int ii=d_inidices[d_particle[i].cellIdx+1-1*xMesh].init; ii<d_inidices[d_particle[i].cellIdx+1-1*xMesh].end; ++ii){
                float dist = sqrt(pow(d_particle[ii].x-d_particle[i].x, 2)+
                                  pow(d_particle[ii].y-d_particle[i].y, 2)+
                                  pow(d_particle[ii].z-d_particle[i].z, 2));

                if(d_particle[ii].tag > 0 && d_particle[i].tag < 1){
                    pred_n++;
                    pred_x += (d_particle[i].x-d_particle[ii].x)/dist;
                    pred_y += (d_particle[i].y-d_particle[ii].y)/dist;
                    pred_z += (d_particle[i].z-d_particle[ii].z)/dist;
                }
                else if(dist < sepa_r){
                   sepa_n++;
                   sepa_x += (d_particle[i].x-d_particle[ii].x)/dist;
                   sepa_y += (d_particle[i].y-d_particle[ii].y)/dist;
                   sepa_z += (d_particle[i].z-d_particle[ii].z)/dist;
                }
                else if(dist < alig_r){
                   alig_n++;
                   alig_x += d_particle[ii].vx;
                   alig_y += d_particle[ii].vy;
                   alig_z += d_particle[ii].vz;
                }
                else if(dist < cohe_r){
                   cohe_n++;
                   cohe_x += d_particle[ii].x;
                   cohe_y += d_particle[ii].y;
                   cohe_z += d_particle[ii].z;
                }
            }
        }
        // x-1, y+1
        if(d_particle[i].cellIdx-1+1*xMesh > 0 && d_particle[i].cellIdx-1+1*xMesh < xMesh*yMesh*zMesh){
            for(int ii=d_inidices[d_particle[i].cellIdx-1+1*xMesh].init; ii<d_inidices[d_particle[i].cellIdx-1+1*xMesh].end; ++ii){
                float dist = sqrt(pow(d_particle[ii].x-d_particle[i].x, 2)+
                                  pow(d_particle[ii].y-d_particle[i].y, 2)+
                                  pow(d_particle[ii].z-d_particle[i].z, 2));

                if(d_particle[ii].tag > 0 && d_particle[i].tag < 1){
                    pred_n++;
                    pred_x += (d_particle[i].x-d_particle[ii].x)/dist;
                    pred_y += (d_particle[i].y-d_particle[ii].y)/dist;
                    pred_z += (d_particle[i].z-d_particle[ii].z)/dist;
                }
                else if(dist < sepa_r){
                   sepa_n++;
                   sepa_x += (d_particle[i].x-d_particle[ii].x)/dist;
                   sepa_y += (d_particle[i].y-d_particle[ii].y)/dist;
                   sepa_z += (d_particle[i].z-d_particle[ii].z)/dist;
                }
                else if(dist < alig_r){
                   alig_n++;
                   alig_x += d_particle[ii].vx;
                   alig_y += d_particle[ii].vy;
                   alig_z += d_particle[ii].vz;
                }
                else if(dist < cohe_r){
                   cohe_n++;
                   cohe_x += d_particle[ii].x;
                   cohe_y += d_particle[ii].y;
                   cohe_z += d_particle[ii].z;
                }
            }
        }
        // x-1, y-1
        if(d_particle[i].cellIdx-1-1*xMesh > 0 && d_particle[i].cellIdx-1-1*xMesh < xMesh*yMesh*zMesh){
            for(int ii=d_inidices[d_particle[i].cellIdx-1-1*xMesh].init; ii<d_inidices[d_particle[i].cellIdx-1-1*xMesh].end; ++ii){
                float dist = sqrt(pow(d_particle[ii].x-d_particle[i].x, 2)+
                                  pow(d_particle[ii].y-d_particle[i].y, 2)+
                                  pow(d_particle[ii].z-d_particle[i].z, 2));

                if(d_particle[ii].tag > 0 && d_particle[i].tag < 1){
                    pred_n++;
                    pred_x += (d_particle[i].x-d_particle[ii].x)/dist;
                    pred_y += (d_particle[i].y-d_particle[ii].y)/dist;
                    pred_z += (d_particle[i].z-d_particle[ii].z)/dist;
                }
                else if(dist < sepa_r){
                   sepa_n++;
                   sepa_x += (d_particle[i].x-d_particle[ii].x)/dist;
                   sepa_y += (d_particle[i].y-d_particle[ii].y)/dist;
                   sepa_z += (d_particle[i].z-d_particle[ii].z)/dist;
                }
                else if(dist < alig_r){
                   alig_n++;
                   alig_x += d_particle[ii].vx;
                   alig_y += d_particle[ii].vy;
                   alig_z += d_particle[ii].vz;
                }
                else if(dist < cohe_r){
                   cohe_n++;
                   cohe_x += d_particle[ii].x;
                   cohe_y += d_particle[ii].y;
                   cohe_z += d_particle[ii].z;
                }
            }
        }
        // x+1, y+1, z+1
        if(d_particle[i].cellIdx+1+1*xMesh+1*xMesh*yMesh > 0 && d_particle[i].cellIdx+1+1*xMesh+1*xMesh*yMesh < xMesh*yMesh*zMesh){
            for(int ii=d_inidices[d_particle[i].cellIdx+1+1*xMesh+1*xMesh*yMesh].init; ii<d_inidices[d_particle[i].cellIdx+1+1*xMesh+1*xMesh*yMesh].end; ++ii){
                float dist = sqrt(pow(d_particle[ii].x-d_particle[i].x, 2)+
                                  pow(d_particle[ii].y-d_particle[i].y, 2)+
                                  pow(d_particle[ii].z-d_particle[i].z, 2));

                if(d_particle[ii].tag > 0 && d_particle[i].tag < 1){
                    pred_n++;
                    pred_x += (d_particle[i].x-d_particle[ii].x)/dist;
                    pred_y += (d_particle[i].y-d_particle[ii].y)/dist;
                    pred_z += (d_particle[i].z-d_particle[ii].z)/dist;
                }
                else if(dist < sepa_r){
                   sepa_n++;
                   sepa_x += (d_particle[i].x-d_particle[ii].x)/dist;
                   sepa_y += (d_particle[i].y-d_particle[ii].y)/dist;
                   sepa_z += (d_particle[i].z-d_particle[ii].z)/dist;
                }
                else if(dist < alig_r){
                   alig_n++;
                   alig_x += d_particle[ii].vx;
                   alig_y += d_particle[ii].vy;
                   alig_z += d_particle[ii].vz;
                }
                else if(dist < cohe_r){
                   cohe_n++;
                   cohe_x += d_particle[ii].x;
                   cohe_y += d_particle[ii].y;
                   cohe_z += d_particle[ii].z;
                }
            }
        }
        // x+1, y-1, z+1
        if(d_particle[i].cellIdx+1-1*xMesh+1*xMesh*yMesh > 0 && d_particle[i].cellIdx+1-1*xMesh+1*xMesh*yMesh < xMesh*yMesh*zMesh){
            for(int ii=d_inidices[d_particle[i].cellIdx+1-1*xMesh+1*xMesh*yMesh].init; ii<d_inidices[d_particle[i].cellIdx+1-1*xMesh+1*xMesh*yMesh].end; ++ii){
                float dist = sqrt(pow(d_particle[ii].x-d_particle[i].x, 2)+
                                  pow(d_particle[ii].y-d_particle[i].y, 2)+
                                  pow(d_particle[ii].z-d_particle[i].z, 2));

                if(d_particle[ii].tag > 0 && d_particle[i].tag < 1){
                    pred_n++;
                    pred_x += (d_particle[i].x-d_particle[ii].x)/dist;
                    pred_y += (d_particle[i].y-d_particle[ii].y)/dist;
                    pred_z += (d_particle[i].z-d_particle[ii].z)/dist;
                }
                else if(dist < sepa_r){
                   sepa_n++;
                   sepa_x += (d_particle[i].x-d_particle[ii].x)/dist;
                   sepa_y += (d_particle[i].y-d_particle[ii].y)/dist;
                   sepa_z += (d_particle[i].z-d_particle[ii].z)/dist;
                }
                else if(dist < alig_r){
                   alig_n++;
                   alig_x += d_particle[ii].vx;
                   alig_y += d_particle[ii].vy;
                   alig_z += d_particle[ii].vz;
                }
                else if(dist < cohe_r){
                   cohe_n++;
                   cohe_x += d_particle[ii].x;
                   cohe_y += d_particle[ii].y;
                   cohe_z += d_particle[ii].z;
                }
            }
        }
        // x-1, y+1, z+1
        if(d_particle[i].cellIdx-1+1*xMesh+1*xMesh*yMesh > 0 && d_particle[i].cellIdx-1+1*xMesh+1*xMesh*yMesh < xMesh*yMesh*zMesh){
            for(int ii=d_inidices[d_particle[i].cellIdx-1+1*xMesh+1*xMesh*yMesh].init; ii<d_inidices[d_particle[i].cellIdx-1+1*xMesh+1*xMesh*yMesh].end; ++ii){
                float dist = sqrt(pow(d_particle[ii].x-d_particle[i].x, 2)+
                                  pow(d_particle[ii].y-d_particle[i].y, 2)+
                                  pow(d_particle[ii].z-d_particle[i].z, 2));

                if(d_particle[ii].tag > 0 && d_particle[i].tag < 1){
                    pred_n++;
                    pred_x += (d_particle[i].x-d_particle[ii].x)/dist;
                    pred_y += (d_particle[i].y-d_particle[ii].y)/dist;
                    pred_z += (d_particle[i].z-d_particle[ii].z)/dist;
                }
                else if(dist < sepa_r){
                   sepa_n++;
                   sepa_x += (d_particle[i].x-d_particle[ii].x)/dist;
                   sepa_y += (d_particle[i].y-d_particle[ii].y)/dist;
                   sepa_z += (d_particle[i].z-d_particle[ii].z)/dist;
                }
                else if(dist < alig_r){
                   alig_n++;
                   alig_x += d_particle[ii].vx;
                   alig_y += d_particle[ii].vy;
                   alig_z += d_particle[ii].vz;
                }
                else if(dist < cohe_r){
                   cohe_n++;
                   cohe_x += d_particle[ii].x;
                   cohe_y += d_particle[ii].y;
                   cohe_z += d_particle[ii].z;
                }
            }
        }
        // x-1, y-1, z+1
        if(d_particle[i].cellIdx-1-1*xMesh+1*xMesh*yMesh > 0 && d_particle[i].cellIdx-1-1*xMesh+1*xMesh*yMesh < xMesh*yMesh*zMesh){
            for(int ii=d_inidices[d_particle[i].cellIdx-1-1*xMesh+1*xMesh*yMesh].init; ii<d_inidices[d_particle[i].cellIdx-1-1*xMesh+1*xMesh*yMesh].end; ++ii){
                float dist = sqrt(pow(d_particle[ii].x-d_particle[i].x, 2)+
                                  pow(d_particle[ii].y-d_particle[i].y, 2)+
                                  pow(d_particle[ii].z-d_particle[i].z, 2));

                if(d_particle[ii].tag > 0 && d_particle[i].tag < 1){
                    pred_n++;
                    pred_x += (d_particle[i].x-d_particle[ii].x)/dist;
                    pred_y += (d_particle[i].y-d_particle[ii].y)/dist;
                    pred_z += (d_particle[i].z-d_particle[ii].z)/dist;
                }
                else if(dist < sepa_r){
                   sepa_n++;
                   sepa_x += (d_particle[i].x-d_particle[ii].x)/dist;
                   sepa_y += (d_particle[i].y-d_particle[ii].y)/dist;
                   sepa_z += (d_particle[i].z-d_particle[ii].z)/dist;
                }
                else if(dist < alig_r){
                   alig_n++;
                   alig_x += d_particle[ii].vx;
                   alig_y += d_particle[ii].vy;
                   alig_z += d_particle[ii].vz;
                }
                else if(dist < cohe_r){
                   cohe_n++;
                   cohe_x += d_particle[ii].x;
                   cohe_y += d_particle[ii].y;
                   cohe_z += d_particle[ii].z;
                }
            }
        }
        // x+1, y+1, z-1
        if(d_particle[i].cellIdx+1+1*xMesh-1*xMesh*yMesh > 0 && d_particle[i].cellIdx+1+1*xMesh-1*xMesh*yMesh < xMesh*yMesh*zMesh){
            for(int ii=d_inidices[d_particle[i].cellIdx+1+1*xMesh-1*xMesh*yMesh].init; ii<d_inidices[d_particle[i].cellIdx+1+1*xMesh-1*xMesh*yMesh].end; ++ii){
                float dist = sqrt(pow(d_particle[ii].x-d_particle[i].x, 2)+
                                  pow(d_particle[ii].y-d_particle[i].y, 2)+
                                  pow(d_particle[ii].z-d_particle[i].z, 2));

                if(d_particle[ii].tag > 0 && d_particle[i].tag < 1){
                    pred_n++;
                    pred_x += (d_particle[i].x-d_particle[ii].x)/dist;
                    pred_y += (d_particle[i].y-d_particle[ii].y)/dist;
                    pred_z += (d_particle[i].z-d_particle[ii].z)/dist;
                }
                else if(dist < sepa_r){
                   sepa_n++;
                   sepa_x += (d_particle[i].x-d_particle[ii].x)/dist;
                   sepa_y += (d_particle[i].y-d_particle[ii].y)/dist;
                   sepa_z += (d_particle[i].z-d_particle[ii].z)/dist;
                }
                else if(dist < alig_r){
                   alig_n++;
                   alig_x += d_particle[ii].vx;
                   alig_y += d_particle[ii].vy;
                   alig_z += d_particle[ii].vz;
                }
                else if(dist < cohe_r){
                   cohe_n++;
                   cohe_x += d_particle[ii].x;
                   cohe_y += d_particle[ii].y;
                   cohe_z += d_particle[ii].z;
                }
            }
        }
        // x+1, y-1, z-1
        if(d_particle[i].cellIdx+1-1*xMesh-1*xMesh*yMesh > 0 && d_particle[i].cellIdx+1-1*xMesh-1*xMesh*yMesh < xMesh*yMesh*zMesh){
            for(int ii=d_inidices[d_particle[i].cellIdx+1-1*xMesh-1*xMesh*yMesh].init; ii<d_inidices[d_particle[i].cellIdx+1-1*xMesh-1*xMesh*yMesh].end; ++ii){
                float dist = sqrt(pow(d_particle[ii].x-d_particle[i].x, 2)+
                                  pow(d_particle[ii].y-d_particle[i].y, 2)+
                                  pow(d_particle[ii].z-d_particle[i].z, 2));

                if(d_particle[ii].tag > 0 && d_particle[i].tag < 1){
                    pred_n++;
                    pred_x += (d_particle[i].x-d_particle[ii].x)/dist;
                    pred_y += (d_particle[i].y-d_particle[ii].y)/dist;
                    pred_z += (d_particle[i].z-d_particle[ii].z)/dist;
                }
                else if(dist < sepa_r){
                   sepa_n++;
                   sepa_x += (d_particle[i].x-d_particle[ii].x)/dist;
                   sepa_y += (d_particle[i].y-d_particle[ii].y)/dist;
                   sepa_z += (d_particle[i].z-d_particle[ii].z)/dist;
                }
                else if(dist < alig_r){
                   alig_n++;
                   alig_x += d_particle[ii].vx;
                   alig_y += d_particle[ii].vy;
                   alig_z += d_particle[ii].vz;
                }
                else if(dist < cohe_r){
                   cohe_n++;
                   cohe_x += d_particle[ii].x;
                   cohe_y += d_particle[ii].y;
                   cohe_z += d_particle[ii].z;
                }
            }
        }
        // x-1, y+1, z-1
        if(d_particle[i].cellIdx-1+1*xMesh-1*xMesh*yMesh > 0 && d_particle[i].cellIdx-1+1*xMesh-1*xMesh*yMesh < xMesh*yMesh*zMesh){
            for(int ii=d_inidices[d_particle[i].cellIdx-1+1*xMesh-1*xMesh*yMesh].init; ii<d_inidices[d_particle[i].cellIdx-1+1*xMesh-1*xMesh*yMesh].end; ++ii){
                float dist = sqrt(pow(d_particle[ii].x-d_particle[i].x, 2)+
                                  pow(d_particle[ii].y-d_particle[i].y, 2)+
                                  pow(d_particle[ii].z-d_particle[i].z, 2));

                if(d_particle[ii].tag > 0 && d_particle[i].tag < 1){
                    pred_n++;
                    pred_x += (d_particle[i].x-d_particle[ii].x)/dist;
                    pred_y += (d_particle[i].y-d_particle[ii].y)/dist;
                    pred_z += (d_particle[i].z-d_particle[ii].z)/dist;
                }
                else if(dist < sepa_r){
                   sepa_n++;
                   sepa_x += (d_particle[i].x-d_particle[ii].x)/dist;
                   sepa_y += (d_particle[i].y-d_particle[ii].y)/dist;
                   sepa_z += (d_particle[i].z-d_particle[ii].z)/dist;
                }
                else if(dist < alig_r){
                   alig_n++;
                   alig_x += d_particle[ii].vx;
                   alig_y += d_particle[ii].vy;
                   alig_z += d_particle[ii].vz;
                }
                else if(dist < cohe_r){
                   cohe_n++;
                   cohe_x += d_particle[ii].x;
                   cohe_y += d_particle[ii].y;
                   cohe_z += d_particle[ii].z;
                }
            }
        }
        // x-1, y-1, z-1
        if(d_particle[i].cellIdx-1-1*xMesh-1*xMesh*yMesh > 0 && d_particle[i].cellIdx-1-1*xMesh-1*xMesh*yMesh < xMesh*yMesh*zMesh){
            for(int ii=d_inidices[d_particle[i].cellIdx-1-1*xMesh-1*xMesh*yMesh].init; ii<d_inidices[d_particle[i].cellIdx-1-1*xMesh-1*xMesh*yMesh].end; ++ii){
                float dist = sqrt(pow(d_particle[ii].x-d_particle[i].x, 2)+
                                  pow(d_particle[ii].y-d_particle[i].y, 2)+
                                  pow(d_particle[ii].z-d_particle[i].z, 2));

                if(d_particle[ii].tag > 0 && d_particle[i].tag < 1){
                    pred_n++;
                    pred_x += (d_particle[i].x-d_particle[ii].x)/dist;
                    pred_y += (d_particle[i].y-d_particle[ii].y)/dist;
                    pred_z += (d_particle[i].z-d_particle[ii].z)/dist;
                }
                else if(dist < sepa_r){
                   sepa_n++;
                   sepa_x += (d_particle[i].x-d_particle[ii].x)/dist;
                   sepa_y += (d_particle[i].y-d_particle[ii].y)/dist;
                   sepa_z += (d_particle[i].z-d_particle[ii].z)/dist;
                }
                else if(dist < alig_r){
                   alig_n++;
                   alig_x += d_particle[ii].vx;
                   alig_y += d_particle[ii].vy;
                   alig_z += d_particle[ii].vz;
                }
                else if(dist < cohe_r){
                   cohe_n++;
                   cohe_x += d_particle[ii].x;
                   cohe_y += d_particle[ii].y;
                   cohe_z += d_particle[ii].z;
                }
            }
        }
        // x+1, y, z+1
        if(d_particle[i].cellIdx+1+1*xMesh*yMesh > 0 && d_particle[i].cellIdx+1+1*xMesh*yMesh < xMesh*yMesh*zMesh){
            for(int ii=d_inidices[d_particle[i].cellIdx+1+1*xMesh*yMesh].init; ii<d_inidices[d_particle[i].cellIdx+1+1*xMesh*yMesh].end; ++ii){
                float dist = sqrt(pow(d_particle[ii].x-d_particle[i].x, 2)+
                                  pow(d_particle[ii].y-d_particle[i].y, 2)+
                                  pow(d_particle[ii].z-d_particle[i].z, 2));

                if(d_particle[ii].tag > 0 && d_particle[i].tag < 1){
                    pred_n++;
                    pred_x += (d_particle[i].x-d_particle[ii].x)/dist;
                    pred_y += (d_particle[i].y-d_particle[ii].y)/dist;
                    pred_z += (d_particle[i].z-d_particle[ii].z)/dist;
                }
                else if(dist < sepa_r){
                   sepa_n++;
                   sepa_x += (d_particle[i].x-d_particle[ii].x)/dist;
                   sepa_y += (d_particle[i].y-d_particle[ii].y)/dist;
                   sepa_z += (d_particle[i].z-d_particle[ii].z)/dist;
                }
                else if(dist < alig_r){
                   alig_n++;
                   alig_x += d_particle[ii].vx;
                   alig_y += d_particle[ii].vy;
                   alig_z += d_particle[ii].vz;
                }
                else if(dist < cohe_r){
                   cohe_n++;
                   cohe_x += d_particle[ii].x;
                   cohe_y += d_particle[ii].y;
                   cohe_z += d_particle[ii].z;
                }
            }
        }
        // x-1, y, z+1
        if(d_particle[i].cellIdx-1+1*xMesh*yMesh > 0 && d_particle[i].cellIdx-1+1*xMesh*yMesh < xMesh*yMesh*zMesh){
            for(int ii=d_inidices[d_particle[i].cellIdx-1+1*xMesh*yMesh].init; ii<d_inidices[d_particle[i].cellIdx-1+1*xMesh*yMesh].end; ++ii){
                float dist = sqrt(pow(d_particle[ii].x-d_particle[i].x, 2)+
                                  pow(d_particle[ii].y-d_particle[i].y, 2)+
                                  pow(d_particle[ii].z-d_particle[i].z, 2));

                if(d_particle[ii].tag > 0 && d_particle[i].tag < 1){
                    pred_n++;
                    pred_x += (d_particle[i].x-d_particle[ii].x)/dist;
                    pred_y += (d_particle[i].y-d_particle[ii].y)/dist;
                    pred_z += (d_particle[i].z-d_particle[ii].z)/dist;
                }
                else if(dist < sepa_r){
                   sepa_n++;
                   sepa_x += (d_particle[i].x-d_particle[ii].x)/dist;
                   sepa_y += (d_particle[i].y-d_particle[ii].y)/dist;
                   sepa_z += (d_particle[i].z-d_particle[ii].z)/dist;
                }
                else if(dist < alig_r){
                   alig_n++;
                   alig_x += d_particle[ii].vx;
                   alig_y += d_particle[ii].vy;
                   alig_z += d_particle[ii].vz;
                }
                else if(dist < cohe_r){
                   cohe_n++;
                   cohe_x += d_particle[ii].x;
                   cohe_y += d_particle[ii].y;
                   cohe_z += d_particle[ii].z;
                }
            }
        }
        // x, y+1, z+1
        if(d_particle[i].cellIdx+1*xMesh+1*xMesh*yMesh > 0 && d_particle[i].cellIdx+1*xMesh+1*xMesh*yMesh < xMesh*yMesh*zMesh){
            for(int ii=d_inidices[d_particle[i].cellIdx+1*xMesh+1*xMesh*yMesh].init; ii<d_inidices[d_particle[i].cellIdx+1*xMesh+1*xMesh*yMesh].end; ++ii){
                float dist = sqrt(pow(d_particle[ii].x-d_particle[i].x, 2)+
                                  pow(d_particle[ii].y-d_particle[i].y, 2)+
                                  pow(d_particle[ii].z-d_particle[i].z, 2));

                if(d_particle[ii].tag > 0 && d_particle[i].tag < 1){
                    pred_n++;
                    pred_x += (d_particle[i].x-d_particle[ii].x)/dist;
                    pred_y += (d_particle[i].y-d_particle[ii].y)/dist;
                    pred_z += (d_particle[i].z-d_particle[ii].z)/dist;
                }
                else if(dist < sepa_r){
                   sepa_n++;
                   sepa_x += (d_particle[i].x-d_particle[ii].x)/dist;
                   sepa_y += (d_particle[i].y-d_particle[ii].y)/dist;
                   sepa_z += (d_particle[i].z-d_particle[ii].z)/dist;
                }
                else if(dist < alig_r){
                   alig_n++;
                   alig_x += d_particle[ii].vx;
                   alig_y += d_particle[ii].vy;
                   alig_z += d_particle[ii].vz;
                }
                else if(dist < cohe_r){
                   cohe_n++;
                   cohe_x += d_particle[ii].x;
                   cohe_y += d_particle[ii].y;
                   cohe_z += d_particle[ii].z;
                }
            }
        }
        // x, y-1, z+1
        if(d_particle[i].cellIdx-1*xMesh+1*xMesh*yMesh > 0 && d_particle[i].cellIdx-1*xMesh+1*xMesh*yMesh < xMesh*yMesh*zMesh){
            for(int ii=d_inidices[d_particle[i].cellIdx-1*xMesh+1*xMesh*yMesh].init; ii<d_inidices[d_particle[i].cellIdx-1*xMesh+1*xMesh*yMesh].end; ++ii){
                float dist = sqrt(pow(d_particle[ii].x-d_particle[i].x, 2)+
                                  pow(d_particle[ii].y-d_particle[i].y, 2)+
                                  pow(d_particle[ii].z-d_particle[i].z, 2));

                if(d_particle[ii].tag > 0 && d_particle[i].tag < 1){
                    pred_n++;
                    pred_x += (d_particle[i].x-d_particle[ii].x)/dist;
                    pred_y += (d_particle[i].y-d_particle[ii].y)/dist;
                    pred_z += (d_particle[i].z-d_particle[ii].z)/dist;
                }
                else if(dist < sepa_r){
                   sepa_n++;
                   sepa_x += (d_particle[i].x-d_particle[ii].x)/dist;
                   sepa_y += (d_particle[i].y-d_particle[ii].y)/dist;
                   sepa_z += (d_particle[i].z-d_particle[ii].z)/dist;
                }
                else if(dist < alig_r){
                   alig_n++;
                   alig_x += d_particle[ii].vx;
                   alig_y += d_particle[ii].vy;
                   alig_z += d_particle[ii].vz;
                }
                else if(dist < cohe_r){
                   cohe_n++;
                   cohe_x += d_particle[ii].x;
                   cohe_y += d_particle[ii].y;
                   cohe_z += d_particle[ii].z;
                }
            }
        }
        // x+1, y, z-1
        if(d_particle[i].cellIdx+1-1*xMesh*yMesh > 0 && d_particle[i].cellIdx+1-1*xMesh*yMesh < xMesh*yMesh*zMesh){
            for(int ii=d_inidices[d_particle[i].cellIdx+1-1*xMesh*yMesh].init; ii<d_inidices[d_particle[i].cellIdx+1-1*xMesh*yMesh].end; ++ii){
                float dist = sqrt(pow(d_particle[ii].x-d_particle[i].x, 2)+
                                  pow(d_particle[ii].y-d_particle[i].y, 2)+
                                  pow(d_particle[ii].z-d_particle[i].z, 2));

                if(d_particle[ii].tag > 0 && d_particle[i].tag < 1){
                    pred_n++;
                    pred_x += (d_particle[i].x-d_particle[ii].x)/dist;
                    pred_y += (d_particle[i].y-d_particle[ii].y)/dist;
                    pred_z += (d_particle[i].z-d_particle[ii].z)/dist;
                }
                else if(dist < sepa_r){
                   sepa_n++;
                   sepa_x += (d_particle[i].x-d_particle[ii].x)/dist;
                   sepa_y += (d_particle[i].y-d_particle[ii].y)/dist;
                   sepa_z += (d_particle[i].z-d_particle[ii].z)/dist;
                }
                else if(dist < alig_r){
                   alig_n++;
                   alig_x += d_particle[ii].vx;
                   alig_y += d_particle[ii].vy;
                   alig_z += d_particle[ii].vz;
                }
                else if(dist < cohe_r){
                   cohe_n++;
                   cohe_x += d_particle[ii].x;
                   cohe_y += d_particle[ii].y;
                   cohe_z += d_particle[ii].z;
                }
            }
        }
        // x-1, y, z-1
        if(d_particle[i].cellIdx-1-1*xMesh*yMesh > 0 && d_particle[i].cellIdx-1-1*xMesh*yMesh < xMesh*yMesh*zMesh){
            for(int ii=d_inidices[d_particle[i].cellIdx-1-1*xMesh*yMesh].init; ii<d_inidices[d_particle[i].cellIdx-1-1*xMesh*yMesh].end; ++ii){
                float dist = sqrt(pow(d_particle[ii].x-d_particle[i].x, 2)+
                                  pow(d_particle[ii].y-d_particle[i].y, 2)+
                                  pow(d_particle[ii].z-d_particle[i].z, 2));

                if(d_particle[ii].tag > 0 && d_particle[i].tag < 1){
                    pred_n++;
                    pred_x += (d_particle[i].x-d_particle[ii].x)/dist;
                    pred_y += (d_particle[i].y-d_particle[ii].y)/dist;
                    pred_z += (d_particle[i].z-d_particle[ii].z)/dist;
                }
                else if(dist < sepa_r){
                   sepa_n++;
                   sepa_x += (d_particle[i].x-d_particle[ii].x)/dist;
                   sepa_y += (d_particle[i].y-d_particle[ii].y)/dist;
                   sepa_z += (d_particle[i].z-d_particle[ii].z)/dist;
                }
                else if(dist < alig_r){
                   alig_n++;
                   alig_x += d_particle[ii].vx;
                   alig_y += d_particle[ii].vy;
                   alig_z += d_particle[ii].vz;
                }
                else if(dist < cohe_r){
                   cohe_n++;
                   cohe_x += d_particle[ii].x;
                   cohe_y += d_particle[ii].y;
                   cohe_z += d_particle[ii].z;
                }
            }
        }
        // x, y+1, z-1
        if(d_particle[i].cellIdx+1*xMesh-1*xMesh*yMesh > 0 && d_particle[i].cellIdx+1*xMesh-1*xMesh*yMesh < xMesh*yMesh*zMesh){
            for(int ii=d_inidices[d_particle[i].cellIdx+1*xMesh-1*xMesh*yMesh].init; ii<d_inidices[d_particle[i].cellIdx+1*xMesh-1*xMesh*yMesh].end; ++ii){
                float dist = sqrt(pow(d_particle[ii].x-d_particle[i].x, 2)+
                                  pow(d_particle[ii].y-d_particle[i].y, 2)+
                                  pow(d_particle[ii].z-d_particle[i].z, 2));

                if(d_particle[ii].tag > 0 && d_particle[i].tag < 1){
                    pred_n++;
                    pred_x += (d_particle[i].x-d_particle[ii].x)/dist;
                    pred_y += (d_particle[i].y-d_particle[ii].y)/dist;
                    pred_z += (d_particle[i].z-d_particle[ii].z)/dist;
                }
                else if(dist < sepa_r){
                   sepa_n++;
                   sepa_x += (d_particle[i].x-d_particle[ii].x)/dist;
                   sepa_y += (d_particle[i].y-d_particle[ii].y)/dist;
                   sepa_z += (d_particle[i].z-d_particle[ii].z)/dist;
                }
                else if(dist < alig_r){
                   alig_n++;
                   alig_x += d_particle[ii].vx;
                   alig_y += d_particle[ii].vy;
                   alig_z += d_particle[ii].vz;
                }
                else if(dist < cohe_r){
                   cohe_n++;
                   cohe_x += d_particle[ii].x;
                   cohe_y += d_particle[ii].y;
                   cohe_z += d_particle[ii].z;
                }
            }
        }
        // x, y-1, z-1
        if(d_particle[i].cellIdx-1*xMesh-1*xMesh*yMesh > 0 && d_particle[i].cellIdx-1*xMesh-1*xMesh*yMesh < xMesh*yMesh*zMesh){
            for(int ii=d_inidices[d_particle[i].cellIdx-1*xMesh-1*xMesh*yMesh].init; ii<d_inidices[d_particle[i].cellIdx-1*xMesh-1*xMesh*yMesh].end; ++ii){
                float dist = sqrt(pow(d_particle[ii].x-d_particle[i].x, 2)+
                                  pow(d_particle[ii].y-d_particle[i].y, 2)+
                                  pow(d_particle[ii].z-d_particle[i].z, 2));

                if(d_particle[ii].tag > 0 && d_particle[i].tag < 1){
                    pred_n++;
                    pred_x += (d_particle[i].x-d_particle[ii].x)/dist;
                    pred_y += (d_particle[i].y-d_particle[ii].y)/dist;
                    pred_z += (d_particle[i].z-d_particle[ii].z)/dist;
                }
                else if(dist < sepa_r){
                   sepa_n++;
                   sepa_x += (d_particle[i].x-d_particle[ii].x)/dist;
                   sepa_y += (d_particle[i].y-d_particle[ii].y)/dist;
                   sepa_z += (d_particle[i].z-d_particle[ii].z)/dist;
                }
                else if(dist < alig_r){
                   alig_n++;
                   alig_x += d_particle[ii].vx;
                   alig_y += d_particle[ii].vy;
                   alig_z += d_particle[ii].vz;
                }
                else if(dist < cohe_r){
                   cohe_n++;
                   cohe_x += d_particle[ii].x;
                   cohe_y += d_particle[ii].y;
                   cohe_z += d_particle[ii].z;
                }
            }
        }

        //Predator
        if(sepa_n > 0){
            float mod = sqrt(pow(sepa_x, 2)+pow(sepa_y, 2)+pow(sepa_z, 2));
            pred_x = pred_x/mod-d_particle[i].vx;
            pred_y = pred_y/mod-d_particle[i].vy;
            pred_z = pred_z/mod-d_particle[i].vz;

            d_calc[i].pred.x = pred_x;
            d_calc[i].pred.y = pred_y;
            d_calc[i].pred.z = pred_z;
        }
        else {
            d_calc[i].pred.x = 0.0;
            d_calc[i].pred.y = 0.0;
            d_calc[i].pred.z = 0.0;
        }

        //Separation
        if(sepa_n > 0){
            float mod = sqrt(pow(sepa_x, 2)+pow(sepa_y, 2)+pow(sepa_z, 2));
            sepa_x = sepa_x/mod-d_particle[i].vx;
            sepa_y = sepa_y/mod-d_particle[i].vy;
            sepa_z = sepa_z/mod-d_particle[i].vz;


            d_calc[i].sepa.x = sepa_x;
            d_calc[i].sepa.y = sepa_y;
            d_calc[i].sepa.z = sepa_z;
        }
        else {
            d_calc[i].sepa.x = 0.0;
            d_calc[i].sepa.y = 0.0;
            d_calc[i].sepa.z = 0.0;
        }

        // Alignment
        if(alig_n > 0){
            float mod = sqrt(pow(alig_x, 2)+pow(alig_y, 2)+pow(alig_z, 2));
            alig_x = alig_x/mod-d_particle[i].vx;
            alig_y = alig_y/mod-d_particle[i].vy;
            alig_z = alig_z/mod-d_particle[i].vz;

            d_calc[i].alig.x = alig_x;
            d_calc[i].alig.y = alig_y;
            d_calc[i].alig.z = alig_z;
        }
        else {
            d_calc[i].alig.x = 0.0;
            d_calc[i].alig.y = 0.0;
            d_calc[i].alig.z = 0.0;
        }

        // Cohesion
        if(cohe_n > 0){
            cohe_x = cohe_x/cohe_n-d_particle[i].x;
            cohe_y = cohe_y/cohe_n-d_particle[i].y;
            cohe_z = cohe_z/cohe_n-d_particle[i].z;

            d_calc[i].cohe.x = cohe_x/sqrt(pow(cohe_x, 2)+pow(cohe_y, 2)+pow(cohe_z, 2));
            d_calc[i].cohe.y = cohe_y/sqrt(pow(cohe_x, 2)+pow(cohe_y, 2)+pow(cohe_z, 2));
            d_calc[i].cohe.z = cohe_z/sqrt(pow(cohe_x, 2)+pow(cohe_y, 2)+pow(cohe_z, 2));
        }
        else {
            d_calc[i].cohe.x = 0.0;
            d_calc[i].cohe.y = 0.0;
            d_calc[i].cohe.z = 0.0;
        }


    }
}

__global__ void integrate_kernel(particle *d_particle, calcStruct *d_calc, int numParticles, float seed){
    unsigned int i = threadIdx.x + blockDim.x * blockIdx.x;
    if(i < numParticles){

        hiprandState_t state;
        hiprand_init(seed,
                    i,
                    0,
                    &state);

        float dt =0.002;
        // dv
        //d_particle[i].theta = d_calc[i].thetaMedium + 0.1*hiprand_normal(&state);
        //d_particle[i].alpha = d_calc[i].alphaMedium + 0.1*hiprand_normal(&state);

        float A = 10.0, B = 10.0, C = 10.0, D = 50.0;

        float dvx = A*d_calc[i].sepa.x+B*d_calc[i].alig.x+C*d_calc[i].cohe.x+D*d_calc[i].pred.x;
        float dvy = A*d_calc[i].sepa.y+B*d_calc[i].alig.y+C*d_calc[i].cohe.y+D*d_calc[i].pred.y;
        float dvz = A*d_calc[i].sepa.z+B*d_calc[i].alig.z+C*d_calc[i].cohe.z+D*d_calc[i].pred.z;

        // v
        float vx = d_particle[i].vx + dvx*dt;
        float vy = d_particle[i].vy + dvy*dt;
        float vz = d_particle[i].vz + dvz*dt;

        d_particle[i].vx = vx;
        d_particle[i].vy = vy;
        d_particle[i].vz = vz;

        // x
        if(d_particle[i].x<0.0 && vx<0.0) {
            d_particle[i].x = 1.0;
            d_particle[i].y = d_particle[i].y+vy*dt;
            d_particle[i].z = d_particle[i].z+vz*dt;
        }
        else if(d_particle[i].x>1.0 && vx > 0.0) {
            //theta = pi-theta;
            d_particle[i].x = 0.0;
            d_particle[i].y = d_particle[i].y+vy*dt;
            d_particle[i].z = d_particle[i].z+vz*dt;
        }
        else if(d_particle[i].y<0.0 && vy < 0.0) {
            //theta = 2.0*pi-theta;
            d_particle[i].x = d_particle[i].x+vx*dt;
            d_particle[i].y = 1.0;
            d_particle[i].z = d_particle[i].z+vz*dt;
        }
        else if(d_particle[i].y>1.0 && vy > 0.0) {
            //theta = 2.0*pi-theta;
            d_particle[i].x = d_particle[i].x+vx*dt;
            d_particle[i].y = 0.0;
            d_particle[i].z = d_particle[i].z+vz*dt;
        }
        else if(d_particle[i].z<0.0 && vz < 0.0) {
            //theta = 2.0*pi-theta;
            d_particle[i].x = d_particle[i].x+vx*dt;
            d_particle[i].y = d_particle[i].y+vy*dt;
            d_particle[i].z = 1.0;
        }
        else if(d_particle[i].z>1.0 && vz > 0.0) {
            d_particle[i].x = d_particle[i].x+vx*dt;
            d_particle[i].y = d_particle[i].y+vy*dt;
            d_particle[i].z = 0.0;
        }
        else {
            d_particle[i].x = d_particle[i].x+vx*dt;
            d_particle[i].y = d_particle[i].y+vy*dt;
            d_particle[i].z = d_particle[i].z+vz*dt;
        }
    }
}

/////////////////////////////  methods ////////////////////////////////////////

SystemSPH::SystemSPH(unsigned int blocks,
                     unsigned int threads,
                     unsigned int xMesh,
                     unsigned int yMesh,
                     unsigned int zMesh) {

    //glewInit();
    //glewExperimental = GL_TRUE;
    this->numBlocks = blocks;
    this->numThreads = threads;
    this->numParticles = blocks*threads;
    this->xMeshDim = xMesh;
    this->yMeshDim = yMesh;
    this->zMeshDim = zMesh;
    this->numIndices = xMesh*yMesh*zMesh;

    this->h_particle = new particle[this->numParticles];
    this->h_inidices = new intvec2[this->numIndices];
    this->h_calc = new calcStruct[this->numParticles];

    this->particleIndices = new GLushort[this->numParticles];

    //init particle values
    this->InitParticleData();

    //Generate openGL buffers
    //vertex array object
    //Create vertex buffer object(s)+
    glGenVertexArrays(1, &this->VAO);
    //Bind the Vertex Array Object first, then bind and set vertex buffer(s) and attribute pointer(s).
    glBindVertexArray(this->VAO);


    // set vertex buffer
    glGenBuffers(1, &this->VBOparticles);
    glBindBuffer(GL_ARRAY_BUFFER, this->VBOparticles);
    glBufferData(GL_ARRAY_BUFFER, this->numParticles*sizeof(particle), this->h_particle, GL_DYNAMIC_COPY); //like cupdamemcy host->device
    hipGraphicsGLRegisterBuffer(&this->cudaResourceBufParticles, this->VBOparticles, hipGraphicsRegisterFlagsNone);

    glGenBuffers(1, &this->EBO);
    glBindBuffer(GL_ELEMENT_ARRAY_BUFFER, this->EBO);
    glBufferData(GL_ELEMENT_ARRAY_BUFFER, this->numParticles*sizeof(GLushort), this->particleIndices, GL_STATIC_DRAW);

    // bind attribute pointer(s)
    // Position attribute
    glVertexAttribPointer(0, 3, GL_FLOAT, GL_FALSE, sizeof(particle), (GLvoid*)(2 * sizeof(GLint)));
    glEnableVertexAttribArray(0);
    // Velocity attribute
    glVertexAttribPointer(1, 3, GL_FLOAT, GL_FALSE, sizeof(particle), (GLvoid*)(2*sizeof(GLint)+3*sizeof(GLfloat)));
    glEnableVertexAttribArray(1);
    // tag attribute
    glVertexAttribPointer(2, 1, GL_INT, GL_FALSE, sizeof(particle), (GLvoid*)(2*sizeof(GLint)+6*sizeof(GLfloat)));
    glEnableVertexAttribArray(2);

    //glBindBuffer(GL_ELEMENT_ARRAY_BUFFER, 0); //unbind EBO
    glBindBuffer(GL_ARRAY_BUFFER, 0); // unbind VBO
    glBindVertexArray(0); // Unbind VAO

    glGenBuffers(1, &this->VBOindices);
    glBindBuffer(GL_ARRAY_BUFFER, this->VBOindices);
    glBufferData(GL_ARRAY_BUFFER, this->numIndices*sizeof(intvec2), this->h_inidices, GL_DYNAMIC_COPY); //like cupdamemcy host->device
    hipGraphicsGLRegisterBuffer(&this->cudaResourceBufIndices, this->VBOindices, hipGraphicsRegisterFlagsNone);


    glGenBuffers(1, &this->VBOcalc);
    glBindBuffer(GL_ARRAY_BUFFER, this->VBOcalc);
    glBufferData(GL_ARRAY_BUFFER, this->numParticles*sizeof(calcStruct), this->h_calc, GL_DYNAMIC_COPY); //like cupdamemcy host->device
    hipGraphicsGLRegisterBuffer(&this->cudaResourceBufCalc, this->VBOcalc, hipGraphicsRegisterFlagsNone);

    /// boundaries
    // allocate memory
    this->boundaryPoints = new point[8];
    this->boundaryIndices = new GLushort[16]; // points per squares dot num of squares
    //create boundaries (init)
    this->CreateBounderiesPoints();
    this->CreateBounderiesIndices();

    //vertex array object
    glGenVertexArrays(1, &this->VAOboundary);
    glBindVertexArray(this->VAOboundary);


    //Create vertex buffer object
    glGenBuffers(1, &this->VBOboundary);
    glBindBuffer(GL_ARRAY_BUFFER, this->VBOboundary);
    glBufferData(GL_ARRAY_BUFFER, sizeof(*this->boundaryPoints)*8, this->boundaryPoints, GL_STATIC_DRAW);

    //Create Element Buffer Objects
    glGenBuffers(1, &this->EBOboundary);
    glBindBuffer(GL_ELEMENT_ARRAY_BUFFER, this->EBOboundary);
	glBufferData(GL_ELEMENT_ARRAY_BUFFER, sizeof(*this->boundaryIndices)*16, this->boundaryIndices, GL_STATIC_DRAW);

    // Position attribute (3D)
    glVertexAttribPointer(0, 3, GL_FLOAT, GL_FALSE, sizeof(*this->boundaryPoints), (GLvoid*)0);
    glEnableVertexAttribArray(0);

    glBindBuffer(GL_ARRAY_BUFFER, 0); // unbind VBOp
    glBindVertexArray(0); // Unbind VAOp
}

SystemSPH::~SystemSPH() {
    //hipFree(d_particle);
    //hipFree(d_inidices);
    glDeleteVertexArrays(1, &this->VAO);
    glDeleteBuffers(1, &this->VBOparticles);
    glDeleteBuffers(1, &this->VBOindices);
    glDeleteBuffers(1, &this->VBOcalc);
    glDeleteBuffers(1, &this->EBO);

    hipGraphicsUnregisterResource(this->cudaResourceBufParticles);
    hipGraphicsUnregisterResource(this->cudaResourceBufIndices);
    hipGraphicsUnregisterResource(this->cudaResourceBufCalc);
}

void SystemSPH::Particle_print() {
    //copy to host
    hipGraphicsMapResources(1, &this->cudaResourceBufParticles, 0);
    size_t size = this->numParticles*sizeof(particle);
    hipGraphicsResourceGetMappedPointer((void **)&this->d_particle, &size, this->cudaResourceBufParticles);
    std::cout << hipMemcpy(this->h_particle, this->d_particle, this->numParticles*sizeof(particle), hipMemcpyDeviceToHost);
    hipGraphicsUnmapResources(1, &this->cudaResourceBufParticles, 0);

    for (int i = 0; i < this->numParticles; ++i) {
        printf("Particle id: %d, particle cell id: %d, position: (%1.3f, %1.3f, %1.3f), velocity:  (%1.3f, %1.3f, %1.3f),",
                this->h_particle[i].id, this->h_particle[i].cellIdx,
                this->h_particle[i].x, this->h_particle[i].y, this->h_particle[i].z,
                this->h_particle[i].vx, this->h_particle[i].vy, this->h_particle[i].vz);
        printf("\n");
    }
}

void SystemSPH::Indices_print() {
    //copy to host
    hipGraphicsMapResources(1, &this->cudaResourceBufIndices, 0);
    size_t size = this->numIndices*sizeof(intvec2);
    hipGraphicsResourceGetMappedPointer((void **)&this->d_inidices, &size, this->cudaResourceBufIndices);
    hipMemcpy(this->h_inidices, this->d_inidices, this->numIndices*sizeof(intvec2), hipMemcpyDeviceToHost);
    hipGraphicsUnmapResources(1, &this->cudaResourceBufIndices, 0);
    //print indices
    for (int i = 0; i < this->numIndices; ++i) {
      printf("Indice pos: %d, init: %d, end: %d, number of particles: %d",
              i, this->h_inidices[i].init, this->h_inidices[i].end,
              this->h_inidices[i].end-this->h_inidices[i].init);
      printf("\n");
    }
}

void SystemSPH::Calc_print() {
    //copy to host
    hipGraphicsMapResources(1, &this->cudaResourceBufCalc, 0);
    size_t size = this->numParticles*sizeof(calcStruct);
    hipGraphicsResourceGetMappedPointer((void **)&this->d_calc, &size, this->cudaResourceBufCalc);
    std::cout << hipMemcpy(this->h_calc, this->d_calc, this->numParticles*sizeof(calcStruct), hipMemcpyDeviceToHost);
    hipGraphicsUnmapResources(1, &this->cudaResourceBufCalc, 0);
    //print indices
    for (int i = 0; i < this->numParticles; ++i) {
      printf("Particle index: %d. Separation = (%1.3f, %1.3f., %1.3f).",
              i, this->h_calc[i].sepa.x, this->h_calc[i].sepa.y, this->h_calc[i].sepa.z);
      printf("\n");
    }
}

void SystemSPH::InitParticleData(){
    srand(time(NULL));
    for (int i = 0; i < this->numParticles; ++i) {
        this->particleIndices[i] = i;
        this->h_particle[i].id = i;
        this->h_particle[i].x = (float)rand()/(float)RAND_MAX;
        this->h_particle[i].y = (float)rand()/(float)RAND_MAX;
        this->h_particle[i].z = (float)rand()/(float)RAND_MAX;
        this->h_particle[i].vx = 1.0*(2.0*(float)rand()/(float)RAND_MAX-1.0);
        this->h_particle[i].vy = 1.0*(2.0*(float)rand()/(float)RAND_MAX-1.0);
        this->h_particle[i].vz = 1.0*(2.0*(float)rand()/(float)RAND_MAX-1.0);
        if(i<25)
            this->h_particle[i].tag = 1;
        else
            this->h_particle[i].tag = 0;
    }
}

void SystemSPH::SortParticles(){

    //get device direction
    //modificate vertex buffers with cuda
    hipGraphicsMapResources(1, &this->cudaResourceBufParticles, 0);
    size_t size = sizeof(particle)*this->numParticles;
    hipGraphicsResourceGetMappedPointer((void **)&this->d_particle, &size, this->cudaResourceBufParticles);

    dim3 blocks(this->numBlocks,1,1);    /* Number of blocks   */
    dim3 threads(this->numThreads,1,1);  /* Number of threads  */

    int j, k;
    /* Major step */
    for (k = 2; k <= this->numParticles; k <<= 1) {
        /* Minor step */
        for (j=k>>1; j>0; j=j>>1) {
                bitonic_sort_kernel<<<blocks, threads>>>(this->d_particle, j, k);
        }
    }
    hipGraphicsUnmapResources(1, &this->cudaResourceBufParticles, 0);
}

void SystemSPH::ClearGridIndices(){

    //get device direction
    //modificate vertex buffers with cuda
    hipGraphicsMapResources(1, &this->cudaResourceBufIndices, 0);
    size_t size = sizeof(intvec2)*this->numIndices;
    hipGraphicsResourceGetMappedPointer((void **)&this->d_inidices, &size, this->cudaResourceBufIndices);

    dim3 blocks(this->numIndices%1024==0
         ? int(this->numIndices/1024) : int(this->numIndices/1024)+1 ,1 ,1);    /* Number of blocks   */
    dim3 threads(1024,1,1);  /* Number of threads  */
    clearGridIndices_kernel<<<blocks, threads>>>(this->d_inidices, this->numParticles);

    hipGraphicsUnmapResources(1, &this->cudaResourceBufIndices, 0);

}

void SystemSPH::BuiltGridIncices(){
    //get device direction
    //Indices
    hipGraphicsMapResources(1, &this->cudaResourceBufIndices, 0);
    size_t sizeIndices = sizeof(intvec2)*this->numIndices;
    hipGraphicsResourceGetMappedPointer((void **)&this->d_inidices, &sizeIndices, this->cudaResourceBufIndices);
    //Paricles
    hipGraphicsMapResources(1, &this->cudaResourceBufParticles, 0);
    size_t sizeParticle = sizeof(particle)*this->numParticles;
    hipGraphicsResourceGetMappedPointer((void **)&this->d_particle, &sizeParticle, this->cudaResourceBufParticles);

    dim3 blocks(this->numBlocks,1,1);    /* Number of blocks   */
    dim3 threads(this->numThreads,1,1);  /* Number of threads  */
    builtGridIncices_kernel<<<blocks, threads>>>(this->d_inidices, this->d_particle, this->numParticles);

    hipGraphicsUnmapResources(1, &this->cudaResourceBufIndices, 0);
    hipGraphicsUnmapResources(1, &this->cudaResourceBufParticles, 0);
}

void SystemSPH::CreateGridIndices(){
    this->ClearGridIndices();
    this->BuiltGridIncices();
}

void SystemSPH::PosParticleCell(){
    //get device direction
    hipGraphicsMapResources(1, &this->cudaResourceBufParticles, 0);
    size_t sizeParticle = sizeof(particle)*this->numParticles;
    hipGraphicsResourceGetMappedPointer((void **)&this->d_particle, &sizeParticle, this->cudaResourceBufParticles);

    dim3 blocks(this->numBlocks,1,1);    /* Number of blocks   */
    dim3 threads(this->numThreads,1,1);  /* Number of threads  */
    posParticleCell_kernel<<<blocks, threads>>>(this->d_particle, this->numParticles,
                                                this->xMeshDim, this->yMeshDim, this->zMeshDim);

    hipGraphicsUnmapResources(1, &this->cudaResourceBufParticles, 0);
}

void SystemSPH::CalcOperations(){
    //get device direction
    //Indices
    hipGraphicsMapResources(1, &this->cudaResourceBufIndices, 0);
    size_t sizeIndices = sizeof(intvec2)*this->numIndices;
    hipGraphicsResourceGetMappedPointer((void **)&this->d_inidices, &sizeIndices, this->cudaResourceBufIndices);
    //Paricles
    hipGraphicsMapResources(1, &this->cudaResourceBufParticles, 0);
    size_t sizeParticle = sizeof(particle)*this->numParticles;
    hipGraphicsResourceGetMappedPointer((void **)&this->d_particle, &sizeParticle, this->cudaResourceBufParticles);
    //calculation
    hipGraphicsMapResources(1, &this->cudaResourceBufCalc, 0);
    size_t sizeCalc = sizeof(calcStruct)*this->numParticles;
    hipGraphicsResourceGetMappedPointer((void **)&this->d_calc, &sizeCalc, this->cudaResourceBufCalc);

    dim3 blocks(this->numBlocks,1,1);    /* Number of blocks   */
    dim3 threads(this->numThreads,1,1);  /* Number of threads  */
    // calculate operations
    calcul_kernel<<<blocks, threads>>>(this->d_particle, this->d_inidices, this->d_calc, this->numParticles,
                                       this->xMeshDim, this->yMeshDim, this->zMeshDim);
    // integrate (Euler Exlicit)
    integrate_kernel<<<blocks, threads>>>(this->d_particle, this->d_calc, this->numParticles, this->seed);

    hipGraphicsUnmapResources(1, &this->cudaResourceBufIndices, 0);
    hipGraphicsUnmapResources(1, &this->cudaResourceBufParticles, 0);
    hipGraphicsUnmapResources(1, &this->cudaResourceBufCalc, 0);
}

void SystemSPH::Calculate(){
    //ind cell pos indes
    this->PosParticleCell();
    //sort particles
    this->SortParticles();
    //create indices
    this->CreateGridIndices();
    //calculation
    this->CalcOperations();

}

void SystemSPH::SeedUpdate(int i){
    this->seed = i;
}

void SystemSPH::Save(const std::string& nameFile){
    //Paricles
    hipGraphicsMapResources(1, &this->cudaResourceBufParticles, 0);
    size_t size = this->numParticles*sizeof(particle);
    hipGraphicsResourceGetMappedPointer((void **)&this->d_particle, &size, this->cudaResourceBufParticles);
    std::cout << hipMemcpy(this->h_particle, this->d_particle, this->numParticles*sizeof(particle), hipMemcpyDeviceToHost);
    hipGraphicsUnmapResources(1, &this->cudaResourceBufParticles, 0);

    std::ofstream ofs (nameFile, std::ofstream::out);
    //heater
    ofs << "Step (or seed) of simulations" << "," << this->seed << "," << "Number of particles" << "," << this->numParticles << std::endl;
    ofs << "PaticleIndice" << "," << "PaticleCell" << "," << "Position_x" << "," << "Position_y" << ","<< "Position_z" << ",";
    ofs << "Velocity_vx" << "," << "Velocity_vy" << ","<< "Velocity_vz" << std::endl;
    //body
    for(int i=0; i<this->numParticles; i++){
        ofs << this->h_particle[i].id << "," << this->h_particle[i].cellIdx << ",";
        ofs << this->h_particle[i].x << "," << this->h_particle[i].y << "," << this->h_particle[i].z << ",";
        ofs << this->h_particle[i].vx << "," << this->h_particle[i].vy << "," << this->h_particle[i].vz << std::endl;
    }
    ofs.close();

    std::cout << "The current data has been saved in: " << nameFile << std::endl;
}




//////////////////////// Plot /////////////////////////////////////////////////
void SystemSPH::DrawParticles(){
    //openGl
    glBindVertexArray(this->VAO);
    //glDrawArrays(GL_POINTS, 0, this->numParticles);
    glDrawElements(GL_POINTS, this->numParticles, GL_UNSIGNED_SHORT, 0);
    glBindVertexArray(0);
}
void SystemSPH::BackGround(float r, float g, float b, float a){
    glClearColor(r, g, b, a);
	glClear(GL_COLOR_BUFFER_BIT | GL_DEPTH_BUFFER_BIT);
}
void SystemSPH::DrawBoundary(){
    //openGl
    glBindVertexArray(this->VAOboundary);
    //glDrawArrays(GL_POINTS, 0, 8);

    glDrawElements(GL_LINE_LOOP, 16, GL_UNSIGNED_SHORT,  0);
    glBindVertexArray(0);
}
void SystemSPH::PolygonMode(){
    //this call will result in wireframe polygons.
    glPolygonMode(GL_FRONT_AND_BACK, GL_LINE);
}

void SystemSPH::CreateBounderiesIndices(){
    // boundaries are drawed using line loop, so:
    this->boundaryIndices[0] = 0;
    this->boundaryIndices[1] = 3;
    this->boundaryIndices[2] = 2;
    this->boundaryIndices[3] = 1;
    this->boundaryIndices[4] = 0;
    this->boundaryIndices[5] = 4;
    this->boundaryIndices[6] = 7;
    this->boundaryIndices[7] = 6;
    this->boundaryIndices[8] = 5;
    this->boundaryIndices[9] = 4;
    this->boundaryIndices[10] = 7;
    this->boundaryIndices[11] = 3;
    this->boundaryIndices[12] = 2;
    this->boundaryIndices[13] = 6;
    this->boundaryIndices[14] = 5;
    this->boundaryIndices[15] = 1;
}

void SystemSPH::CreateBounderiesPoints(){
    //point 0
    this->boundaryPoints[0].x = -1.0;
    this->boundaryPoints[0].y = -1.0;
    this->boundaryPoints[0].z = -1.0;
    //point 1
    this->boundaryPoints[1].x = 1.0;
    this->boundaryPoints[1].y = -1.0;
    this->boundaryPoints[1].z = -1.0;
    //point 2
    this->boundaryPoints[2].x = 1.0;
    this->boundaryPoints[2].y = 1.0;
    this->boundaryPoints[2].z = -1.0;
    //point 3
    this->boundaryPoints[3].x = -1.0;
    this->boundaryPoints[3].y = 1.0;
    this->boundaryPoints[3].z = -1.0;
    //point 4
    this->boundaryPoints[4].x = -1.0;
    this->boundaryPoints[4].y = -1.0;
    this->boundaryPoints[4].z = 1.0;
    //point 5
    this->boundaryPoints[5].x = 1.0;
    this->boundaryPoints[5].y = -1.0;
    this->boundaryPoints[5].z = 1.0;
    //point 6
    this->boundaryPoints[6].x = 1.0;
    this->boundaryPoints[6].y = 1.0;
    this->boundaryPoints[6].z = 1.0;
    //point 7
    this->boundaryPoints[7].x = -1.0;
    this->boundaryPoints[7].y = 1.0;
    this->boundaryPoints[7].z = 1.0;
}
